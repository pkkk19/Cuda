#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>

#include "lodepng.h"

//compile with c++ lodepng file

//nvcc CudaNegative.cu lodepng.cpp

__global__ void square(unsigned char * ImageOuput, unsigned char * Image, int a , int b){

	int x[]={NULL,NULL,NULL,NULL,NULL,NULL,NULL,NULL,NULL};

	int red=0,green=0,blue=0,trans=0;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i==0){
		x[0]=i;
		x[1]=i+1;
		x[2]=i+b;
		x[3]=i+b+1;

	}
	else if(i>0 && i<(b-1)){
		x[0]=i;
		x[1]=i+1;
		x[2]=i-1;
		x[3]=i+b;
		x[4]=1+i+b;
		x[5]=i+b-1;
	}
	else if (i==(b-1)){
		x[0]=i;
		x[1]=i-1;
		x[2]=i+b;
		x[3]=i+b-1;
	}
	else if(((i > b-1 && i< (a*b)-b) && ((i+1) % b ==0))){
		x[0]=i;
		x[1]=i-1;
		x[2]=i-b;
		x[3]=i-b-1;
		x[4]=i+b;
		x[5]=i+b-1;
	}
	else if (i==((a*b)-1)){
		x[0]=i;
		x[1]=i-1;
		x[2]=i-b-1;
		x[3]=i-b;
	}
	else if(i>((a*b)-b) && i < (a*b)){
		x[0]=i;
		x[1]=i+1;
		x[2]=i-1;
		x[3]=i-b;
		x[4]=i-b-1;
		x[5]=i-b+1;
	}
	else if(i==(a*b)-b){
		x[0]=i;
		x[1]=i+1;
		x[2]=i-b;
		x[3]=i-b+1;
	}
	else if((i>b-1 &&i<(a*b)-(2*b+1))&&i % b ==0){
		x[0]=i;
		x[1]=i+1;
		x[2]=i+b;
		x[3]=i+b+1;
		x[4]=i-b;
		x[5]=i-b+1;

	}
	else{
		x[0]=i;
		x[1]=i+1;
		x[2]=i-1;
		x[3]=i+b;
		x[4]=i+b+1;
		x[5]=i+b-1;
		x[6]=i-b;
		x[7]=i-b+1;
		x[8]=i-b-1;
	}
	int pixel = i*4;
	int c=0;
for (int i=0;i<sizeof(x)/sizeof(x[0]);i++){
	if(x[i]!=NULL){
		red+= Image[x[i]*4];
		green+= Image[x[i]*4+1];
		blue+= Image[x[i]*4+2];
		c++;
		}

	}
		red=red/c;
		green=green/c;
		blue=blue/c;
		trans=Image[i*4+3];
		ImageOuput[pixel] = red;
		ImageOuput[1+pixel] = green;
		ImageOuput[2+pixel] = blue;
		ImageOuput[3+pixel] = trans;
}

int main(int argc, char **argv){

	unsigned int error;
	unsigned int encError;
	unsigned char* image;
	unsigned int width;
	unsigned int height;
	const char* filename = "HCK.png";
	const char* newFileName = "generated_cuda.png";

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
	}

	const int ARRAY_SIZE = width*height*4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imageInput[ARRAY_SIZE * 4];
	unsigned char host_imageOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imageInput[i] = image[i];
	}

	// declare GPU memory pointers
	unsigned char * d_in;
	unsigned char * d_out;
	int a=height;//height
	int b=width;//widht

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);
	// hipMalloc((void*) &a, sizeof(int));
	// hipMalloc(( void*) &b, sizeof(int));


	hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);
	// hipMemcpy(a, height, sizeof(int), hipMemcpyHostToDevice);
	// hipMemcpy(b, width, sizeof(int), hipMemcpyHostToDevice);


	// launch the kernel
	square<<<height, width>>>(d_out, d_in,a,b);

	// copy back the result array to the CPU
	hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
	
	encError = lodepng_encode32_file(newFileName, host_imageOutput, width, height);
	if(encError){
		printf("error %u: %s\n", error, lodepng_error_text(encError));
	}
	printf("%d->%d",height, width);
	//free(image);
	//free(host_imageInput);
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
